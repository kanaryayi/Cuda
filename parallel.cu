#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>

#define MESSAGE_SIZE_PER_LINE 4+1
#define BUFSIZE 102
#define NUMBER_OF_LINES 15360
#define THREAD_NUMBER_PER_BLOCK 384
#define BLOCK_NUMBER 50
#define CHARACTER_AMOUNT_THREAD 100

//__device__ char* allMessage;
__device__ int allMessageSize = 0;
__global__ void decode(char* file, char* allMessage) {
	int globalid = blockIdx.x*blockDim.x + threadIdx.x;
	int messageSize = CHARACTER_AMOUNT_THREAD;
	int i = 0;
	int index = messageSize * globalid;
	int bound = index + CHARACTER_AMOUNT_THREAD;
	char* message = (char*) malloc(messageSize);
	for (; index < bound; index++) {
		if (file[index] == ',') {
			message[i++] = file[index+1];
			
		}
	}
	if (i == 0)return;

	for (index = globalid*CHARACTER_AMOUNT_THREAD; index < (globalid*CHARACTER_AMOUNT_THREAD)+i; index++) {

		allMessage[index] = message[index-(globalid*CHARACTER_AMOUNT_THREAD)];

	}

}
char *getFileLines() {
	char *allFile;
	allFile = (char*)malloc((NUMBER_OF_LINES*BUFSIZE)+1);
	FILE *file = fopen("encodedfile.txt", "rb");
	allFile[0] = '\0';
	if (file)
	{
		
		fread(allFile, NUMBER_OF_LINES*BUFSIZE,1,file);
		fclose(file);
	
		allFile[NUMBER_OF_LINES*BUFSIZE] = 0;
		
	}
	
	return allFile;
}
void writeFile(char* message) {
	FILE *file = fopen("decodedfile.txt", "w");
	
	int i;
	for (i = 0; i <  NUMBER_OF_LINES*BUFSIZE; i++) {
		if (message[i]) {
			
			fputc(message[i],file);
			
		}

	}
	
	
	fclose(file);
}
char* startGpuProcess(char * file) {

	int size = strlen(file);
	char* allFilesGpu, *message,*allMessage;
	
	message = (char*)malloc(1+NUMBER_OF_LINES*BUFSIZE);

	hipMalloc((void**)&allFilesGpu, NUMBER_OF_LINES*BUFSIZE);
	hipMalloc((void**)&allMessage, NUMBER_OF_LINES*BUFSIZE);

	clock_t start = clock(); 
	hipMemcpy(allFilesGpu,file, NUMBER_OF_LINES*BUFSIZE,hipMemcpyHostToDevice);
	printf("2\n");
	decode << < BLOCK_NUMBER, THREAD_NUMBER_PER_BLOCK >> >(allFilesGpu,allMessage);
	printf("3\n");
	hipDeviceSynchronize();
	hipMemcpy(message, allMessage, NUMBER_OF_LINES*BUFSIZE, hipMemcpyDeviceToHost);
	printf("4\n");
	message[NUMBER_OF_LINES*BUFSIZE] = '\0';
	
	hipDeviceReset();
	clock_t end = clock();
	clock_t millis = end - start;
	printf("Code executed in %f milliseconds.\n", millis / double(CLOCKS_PER_SEC) * 1000);
	return message;
}
int main(void)
{
	char* secret;
	
	
	secret = startGpuProcess(getFileLines());
	writeFile(secret);
	
}
